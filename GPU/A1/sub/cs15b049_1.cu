
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){

    int num_device;
    hipGetDeviceCount(&num_device);
    
    for(int i=0;i<num_device;i++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);

        FILE* fptr = fopen("cs15b049_1_out.txt","w");

        fprintf(fptr,"%d\n",prop.localL1CacheSupported);
        fprintf(fptr,"%d\n",prop.globalL1CacheSupported);
        fprintf(fptr,"%d\n",prop.l2CacheSize);
        fprintf(fptr,"%d\n",prop.maxThreadsPerBlock);
        fprintf(fptr,"%d\n",prop.regsPerBlock);
        fprintf(fptr,"%d\n",prop.regsPerMultiprocessor);
        fprintf(fptr,"%d\n",prop.warpSize);
        fprintf(fptr,"%zu\n",prop.totalGlobalMem);

    }

    return 0;
}