
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


//device kernel code
__global__ void vec_add( int* a, int* b,int* c,int N){

    int i = ( blockIdx.x * blockDim.x )+ threadIdx.x;

    if(i<N)
        c[i]= a[i]+b[i];

}

int main(int argc , char* argv[]){
    FILE* f1;
    FILE* f2;
    if(argc>2){
        f1= fopen(argv[1],"r");
        f2= fopen(argv[2],"r");
    }else
        return 0;
    

    int N = 32768;
    // int N = 5;
    int size = sizeof(int)*N;
    int* h_A = (int*)malloc(size);
    int* h_B = (int*)malloc(size);
    int* h_C = (int*)malloc(size);

    int i=0;
    for(i=0;i<N;i++){
        fscanf(f1,"%d\n",&h_A[i]);
        fscanf(f2,"%d\n",&h_B[i]);
    }

    // Allocate vectors in device memory
    int* d_A;
    hipMalloc(&d_A,size);
    int* d_B;
    hipMalloc(&d_B,size);
    int* d_C;
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    // set kernel prams
    int threads_per_block = 256;
    int blocks_per_grid = 128;
    
    vec_add<<<blocks_per_grid,threads_per_block>>>(d_A,d_B,d_C,N);

    //copy result array back to host
    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    FILE* out = fopen("cs15b049_3_out.txt","w");
    for(int i=0;i<N;i++){
        fprintf(out,"%d %d %d\n",h_A[i],h_B[i],h_C[i]);
    }

}