#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


//device kernel code
__global__ void vec_add( int* a, int* b,int* c,int N,int ops_per_thread){

    int idx = ( blockIdx.x * blockDim.x )+ threadIdx.x;

    for(int i=ops_per_thread*idx; i< ops_per_thread*(idx+1);i++){
        if(i<N)
            c[i]= a[i]+b[i];
    }

}

int main(int argc , char* argv[]){
    FILE* f1;
    FILE* f2;

      // set kernel prams
      int threads_per_block;
      int ops_per_thread;
      int N;
        

    if(argc>5){
        threads_per_block = atoi(argv[1]);
        ops_per_thread = atoi(argv[2]);
        N = atoi(argv[3]);
        f1= fopen(argv[4],"r");
        f2= fopen(argv[5],"r");

    }else
        return 0;
    

    int blocks_per_grid =  (N + (threads_per_block*ops_per_thread)-1)/(threads_per_block*ops_per_thread) ;

    // int N = 5;
    int size = sizeof(int)*N;
    int* h_A = (int*)malloc(size);
    int* h_B = (int*)malloc(size);
    int* h_C = (int*)malloc(size);

    //clock variable for measuring runtime
    clock_t begin, end;
	double timeSpent;

    //take input from file
    int i=0;
    for(i=0;i<N;i++){
        fscanf(f1,"%d\n",&h_A[i]);
        fscanf(f2,"%d\n",&h_B[i]);
    }

    // Allocate vectors in device memory
    int* d_A;
    hipMalloc(&d_A,size);
    int* d_B;
    hipMalloc(&d_B,size);
    int* d_C;
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

  
    //start time
    begin  = clock();
    //invoke kernel
    vec_add<<<blocks_per_grid,threads_per_block>>>(d_A,d_B,d_C,N,ops_per_thread);
    //synchronise
    hipDeviceSynchronize();
    //end time
    end = clock();

    timeSpent = (double)(end - begin) * 1000 / CLOCKS_PER_SEC;	// milliseconds

    //copy result array back to host
    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("%lf\n",timeSpent);
}