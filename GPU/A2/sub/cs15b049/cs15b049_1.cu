#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
        for(unsigned j=0; j < numCols; j++){
             mat[i*numCols + j] = i*2.1f + j*3.2f;
        }
}

void print_matrix_to_file(double *mat,unsigned numRows,unsigned numCols){
    const char *fname = "assignment2_out";
    FILE *f = fopen(fname, "a");
 
    for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
            fprintf(f,"%4.4f ", mat[i*numCols + j]);
        
            fprintf(f,"\n");
    }
    fclose(f);
}

__global__ void MatrixMulKernelA(double* M, double* N, double* P, int Width) {
    
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    
    if ((Row < Width) && (Col < Width)) {
        double Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
        P[Row*Width+Col] = Pvalue;
    }

}



__global__ void MatrixMulKernelB(double* M, double* N, double* P, int Width) {
    
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    
    if ((Row < Width) && (Col < Width)) {
        double Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
        P[Row*Width+Col] = Pvalue;
    }

}


int main(){
    int N = 8192;
    int size =  sizeof(double)*N*N;
    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    fill_matrix(h_A,N,N);
    fill_matrix(h_B,N,N);

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    dim3 threads(16,16);
    dim3 blocks(512,512);

    MatrixMulKernelA<<<blocks,threads>>>(d_A,d_B,d_C,N);

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

    print_matrix_to_file(h_C,N,N);

    MatrixMulKernelB<<<blocks,threads>>>(d_A,d_B,d_C,N);

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

    print_matrix_to_file(h_C,N,N);
}