#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
        for(unsigned j=0; j < numCols; j++){
             mat[i*numCols + j] = i*2.1f + j*3.2f;
        }
}

void print_matrix_to_file(double *mat,unsigned numRows,unsigned numCols){
    const char *fname = "assignment2_out";
    FILE *f = fopen(fname, "w");
 
    for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
            fprintf(f,"%4.4f ", mat[i*numCols + j]);
        
            fprintf(f,"\n");
    }
    fclose(f);
}

__global__ void MatrixMulKernel(double* M, double* N, double* P, int M_W,int N_W,int K_W) {
    
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    
    if ((Row < M_W) && (Col < K_W)) {
        double Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < N_W; ++k) {
            Pvalue += M[Row*N_W+k]*N[k*K_W+Col];
        }
        P[Row*K_W+Col] = Pvalue;
    }

}


int main(){
    int M = 4096;
    int N = 8192;
    int K = 16384;
    int size_A =  sizeof(double)*M*N;
    int size_B =  sizeof(double)*N*K;
    int size_C =  sizeof(double)*M*K;
    double* h_A = (double*)malloc(size_A);
    double* h_B = (double*)malloc(size_B);
    double* h_C = (double*)malloc(size_C);

    fill_matrix(h_A,M,N);
    fill_matrix(h_B,N,K);

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A,size_A);
    hipMalloc(&d_B,size_B);
    hipMalloc(&d_C,size_C);

    //copy to device
    hipMemcpy(d_A,h_A,size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size_B,hipMemcpyHostToDevice);

    int thread_x = 16;
    dim3 threads(thread_x,thread_x);
    int block_x = (M + thread_x -1)/thread_x;
    int block_y = (K + thread_x -1)/thread_x;
    dim3 blocks(block_y,block_x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,M,N,K);

    hipEventRecord(stop);

    hipMemcpy(h_C,d_C,size_C,hipMemcpyDeviceToHost);


    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); 
    printf("%lf\n",milliseconds);

    print_matrix_to_file(h_C,M,K);

}