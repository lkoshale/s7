#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define TILE_WIDTH_A 4
#define TILE_WIDTH_B 8
#define TILE_WIDTH_C 16
#define TILE_WIDTH_D 32

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
        for(unsigned j=0; j < numCols; j++){
             mat[i*numCols + j] = i*2.1f + j*3.2f;
        }
}

void print_matrix_to_file(double *mat,unsigned numRows,unsigned numCols){
    const char *fname = "assignment2_out";
    FILE *f = fopen(fname, "a");
 
    for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
            fprintf(f,"%4.4f ", mat[i*numCols + j]);
        
            fprintf(f,"\n");
    }
    fclose(f);
}


//using tiling
__global__ void KernelA(double* M, double* N, double* P, int Width){
    
    __shared__ double ds_M[TILE_WIDTH_A][TILE_WIDTH_A];
    __shared__ double ds_N[TILE_WIDTH_A][TILE_WIDTH_A];


    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * blockDim.y+ty; 
    int Col = bx * blockDim.x+tx;

    double Pvalue = 0;

    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < Width/TILE_WIDTH_A; ++p) {
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row*Width + (p*TILE_WIDTH_A)+tx];
        ds_N[ty][tx] = N[((p*TILE_WIDTH_A)+ty)*Width + Col];
        
        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH_A; ++i)
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
        __syncthreads();
    }
    P[Row*Width+Col] = Pvalue;

}

__global__ void KernelB(double* M, double* N, double* P, int Width){
    
    __shared__ double ds_M[TILE_WIDTH_B][TILE_WIDTH_B];
    __shared__ double ds_N[TILE_WIDTH_B][TILE_WIDTH_B];


    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * blockDim.y+ty; 
    int Col = bx * blockDim.x+tx;

    double Pvalue = 0;

    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < Width/TILE_WIDTH_B; ++p) {
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH_B+tx];
        ds_N[ty][tx] = N[(p*TILE_WIDTH_B+ty)*Width + Col];
        
        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH_B; ++i)
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
        __syncthreads();
    }
    P[Row*Width+Col] = Pvalue;

}

__global__ void KernelC(double* M, double* N, double* P, int Width){
    
    __shared__ double ds_M[TILE_WIDTH_C][TILE_WIDTH_C];
    __shared__ double ds_N[TILE_WIDTH_C][TILE_WIDTH_C];


    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * blockDim.y+ty; 
    int Col = bx * blockDim.x+tx;

    double Pvalue = 0;

    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < Width/TILE_WIDTH_C; ++p) {
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH_C+tx];
        ds_N[ty][tx] = N[(p*TILE_WIDTH_C+ty)*Width + Col];
        
        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH_C; ++i)
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
        __syncthreads();
    }
    P[Row*Width+Col] = Pvalue;

}

__global__ void KernelD(double* M, double* N, double* P, int Width){
    
    __shared__ double ds_M[TILE_WIDTH_D][TILE_WIDTH_D];
    __shared__ double ds_N[TILE_WIDTH_D][TILE_WIDTH_D];


    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * blockDim.y+ty; 
    int Col = bx * blockDim.x+tx;

    double Pvalue = 0;

    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < Width/TILE_WIDTH_D; ++p) {
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH_D+tx];
        ds_N[ty][tx] = N[(p*TILE_WIDTH_D+ty)*Width + Col];
        
        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH_D; ++i)
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
        __syncthreads();
    }
    P[Row*Width+Col] = Pvalue;

}


int main(){
    int N = 8192;
    int size =  sizeof(double)*N*N;
    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    hipEvent_t startA, stopA;
    hipEvent_t startB, stopB;
    hipEvent_t startC, stopC;
    hipEvent_t startD, stopD;

    fill_matrix(h_A,N,N);
    fill_matrix(h_B,N,N);

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    {
        dim3 threads(TILE_WIDTH_A,TILE_WIDTH_A);
        int b_wid = (N + TILE_WIDTH_A - 1)/TILE_WIDTH_A ;
        dim3 blocks(b_wid,b_wid);
        hipEventCreate(&startA);
        hipEventCreate(&stopA);

        hipEventRecord(startA);

        KernelA<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stopA);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

        hipEventSynchronize(stopA);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startA, stopA); 
        printf("%lf\n",milliseconds);

        print_matrix_to_file(h_C,N,N);

    }

    {
        dim3 threads(TILE_WIDTH_B,TILE_WIDTH_B);
        int b_wid = (N + TILE_WIDTH_B - 1)/TILE_WIDTH_B ;
        dim3 blocks(b_wid,b_wid);
        hipEventCreate(&startB);
        hipEventCreate(&stopB);

        hipEventRecord(startB);

        KernelB<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stopB);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

        hipEventSynchronize(stopB);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startB, stopB); 
        printf("%lf\n",milliseconds);

        print_matrix_to_file(h_C,N,N);

    }

    {
        dim3 threads(TILE_WIDTH_C,TILE_WIDTH_C);
        int b_wid = (N + TILE_WIDTH_C - 1)/TILE_WIDTH_C ;
        dim3 blocks(b_wid,b_wid);
        hipEventCreate(&startC);
        hipEventCreate(&stopC);

        hipEventRecord(startC);

        KernelC<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stopC);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

        hipEventSynchronize(stopC);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startC, stopC); 
        printf("%lf\n",milliseconds);

        print_matrix_to_file(h_C,N,N);

    }

    {
        dim3 threads(TILE_WIDTH_D,TILE_WIDTH_D);
        int b_wid = (N + TILE_WIDTH_D - 1)/TILE_WIDTH_D ;
        dim3 blocks(b_wid,b_wid);
        hipEventCreate(&startD);
        hipEventCreate(&stopD);

        hipEventRecord(startD);

        KernelD<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stopD);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

        hipEventSynchronize(stopD);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startD, stopD); 
        printf("%lf\n",milliseconds);

        print_matrix_to_file(h_C,N,N);

    }
}