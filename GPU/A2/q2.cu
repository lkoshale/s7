#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<string.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
        for(unsigned j=0; j < numCols; j++){
             mat[i*numCols + j] = i*2.1f + j*3.2f;
        }
}

void print_matrix_to_file(double *mat,unsigned numRows,unsigned numCols){
    const char *fname = "assignment2_out";
    FILE *f = fopen(fname, "a");
 
    for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
            fprintf(f,"%4.4f ", mat[i*numCols + j]);
        
            fprintf(f,"\n");
    }
    fclose(f);
}

__global__ void MatrixMulKernel(double* M, double* N, double* P, int Width) {
    
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    
    if ((Row < Width) && (Col < Width)) {
        double Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row*Width+k]*N[k*Width+Col];
        }
        P[Row*Width+Col] = Pvalue;
    }

}


int main(int argc,char* argv[]){

    int N = 8192;
    int size =  sizeof(double)*N*N;
    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    fill_matrix(h_A,N,N);
    fill_matrix(h_B,N,N);

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);


    {
        dim3 threads(2,2);
        dim3 blocks(4096,4096);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }

    {
        dim3 threads(4,4);
        dim3 blocks(2048,2048);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }

    {
        dim3 threads(8,8);
        dim3 blocks(1024,1024);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }

    {
        dim3 threads(8,16);
        dim3 blocks(1024,512);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }



    //8 data points for diiferent blcok configurations
    {
        dim3 threads(16,16);
        dim3 blocks(512,512);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }

    {
        dim3 threads(16,32);
        dim3 blocks(512,256);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }

    {
        dim3 threads(32,32);
        dim3 blocks(256,256);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

        hipEventRecord(stop);

        hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); 
        printf("%lf\n",milliseconds);
        print_matrix_to_file(h_C,N,N);
    
    }






}
