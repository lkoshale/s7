#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define MAT_SIZE_A 32
#define BLOCK_DIM_A 16


void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
        for(unsigned j=0; j < numCols; j++){
             mat[i*numCols + j] = i*2.1f + j*3.2f;
        }
}

void print_matrix_to_file(double *mat,unsigned numRows,unsigned numCols){
    const char *fname = "assignment2_out";
    FILE *f = fopen(fname, "w");
 
    for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
            fprintf(f,"%4.4f ", mat[i*numCols + j]);
        
            fprintf(f,"\n");
    }
    fclose(f);
}

__global__ void kernelA(double* M, double* N, double* P) {

    __shared__ double ds_M[BLOCK_DIM_A][MAT_SIZE_A];
    __shared__ double ds_N[BLOCK_DIM_A][MAT_SIZE_A];

    int Width = MAT_SIZE_A;
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    
    // Calculate the row index of the P element and M
    int Row = by*blockDim.y+ty;
    // Calculate the column index of P and N
    int Col = bx*blockDim.x+tx;


    int px = Width/blockDim.x;
    int py = Width/blockDim.y;

    if( (Row<Width) && (Col<Width) ){

        //load your part of rows from M
        for(int i=0;i<px;i++){
            ds_M[ty][tx*px+i] = M[(Row*Width)+(tx*px)+i];
        }

        //load your part of cols from N
        //as transpose matrix
        for(int i=0;i<py;i++){
            int idx = (ty*py+i)*Width+Col;
            ds_N[tx][ty*py+i] = N[idx];
        }

        __syncthreads();


        double Pvalue = 0;
        //compute multiplication each thread individually
        // Row*Col'
        for(int i=0 ;i<Width;++i ){
            Pvalue+= ds_M[tx][i] * ds_N[ty][i];
        }
        
        P[Row*Width+Col]=Pvalue;
        
    }
}


int main(){

    int N = MAT_SIZE_A;
    int size =  sizeof(double)*N*N;
    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    fill_matrix(h_A,N,N);
    fill_matrix(h_B,N,N);

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    dim3 threads(BLOCK_DIM_A,BLOCK_DIM_A);

    int block_per_grid = (N +BLOCK_DIM_A -1)/ BLOCK_DIM_A;

    dim3 blocks(block_per_grid,block_per_grid);

    //invoke kernel
    kernelA<<<blocks,threads>>>(d_A,d_B,d_C);

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

    print_matrix_to_file(h_C,N,N);

}