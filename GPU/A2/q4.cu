#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define TILE_WIDTH 16


void fill_matrix(double *mat, unsigned numRows, unsigned numCols)
{
    for(unsigned i=0; i < numRows; i++)
        for(unsigned j=0; j < numCols; j++){
             mat[i*numCols + j] = i*2.1f + j*3.2f;
        }
}

void print_matrix_to_file(double *mat,unsigned numRows,unsigned numCols){
    const char *fname = "assignment2_out";
    FILE *f = fopen(fname, "w");
 
    for(unsigned i=0; i < numRows; i++)
    {
        for(unsigned j=0; j < numCols; j++)
            fprintf(f,"%4.4f ", mat[i*numCols + j]);
        
            fprintf(f,"\n");
    }
    fclose(f);
}


//using tiling
__global__ void MatrixMulKernel(double* M, double* N, double* P, int Width){
    
    __shared__ double ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ double ds_N[TILE_WIDTH][TILE_WIDTH];


    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * blockDim.y+ty; 
    int Col = bx * blockDim.x+tx;

    double Pvalue = 0;

    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < Width/TILE_WIDTH; ++p) {
        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH+tx];
        ds_N[ty][tx] = N[(p*TILE_WIDTH+ty)*Width + Col];
        
        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH; ++i)
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
        __syncthreads();
    }
    P[Row*Width+Col] = Pvalue;

}

int main(){
    int N = 8192;
    int size =  sizeof(double)*N*N;
    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    fill_matrix(h_A,N,N);
    fill_matrix(h_B,N,N);

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);

    //copy to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

    dim3 threads(16,16);
    dim3 blocks(512,512);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    MatrixMulKernel<<<blocks,threads>>>(d_A,d_B,d_C,N);

    hipEventRecord(stop);

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); 
    printf("%lf\n",milliseconds);

    print_matrix_to_file(h_C,N,N);

}