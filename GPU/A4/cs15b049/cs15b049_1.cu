/*
Template code for convolution. CS6023, IITM */
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define W 1024 // Input DIM
#define OW (W-4) // Output DIM
#define D 8   // Input and Kernel Depth
#define T 5  // Kernel DIM
#define N 128 // Number of kernels


#define TILE_WIDTH 32

void fillMatrix(unsigned char *matrix){

unsigned char (*m)[W][D]=(unsigned char (*)[W][D])matrix;

for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			m[i][j][k]=(i*j+j*k+i*k+i*2+j*3+k*4)%255;
				}
			}
		}
}



void fillKernel(float *kernel){

float (*t)[T][T][D]=(float (*)[T][T][D])kernel;

for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
			t[i][j][k][l]=fmod(-(i+1)*2.1+(j+1)*3.2-(k+1)*4.8+(l+1)*7.1,1.0);
				}
			}
		}
	}
}



void print_matrix_to_file(float *m){

	const char *fname = "assignment4_out";
	FILE *f = fopen(fname, "w");

	float (*mat)[OW][OW]=(float (*)[OW][OW])m;		

	for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < OW; j++)
			for(unsigned k=0;k<OW;k++)
				fprintf(f,"%4.4f ", mat[i][j][k]);
		fprintf(f,"\n");
	}
	fclose(f);
}



__global__ void Convolve(unsigned char* mat,float* out,const float*  __restrict__ kern ){

	__shared__ unsigned char M[TILE_WIDTH+4][TILE_WIDTH+4][D];

	unsigned char (*matrix)[W][D]=(unsigned char (*)[W][D])mat;
	float (*kernel)[T][T][D]=(float (*)[T][T][D])kern;
	

	int k_idx = blockIdx.z;

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int x = blockIdx.x*blockDim.x + tx;
	int y = blockIdx.y*blockDim.y + ty;


	//compute out[x,y]
	if(x < OW && y< OW ){
		//load input in shared var
		for(int i=0;i<D;i++)
			M[tx+2][ty+2][i]= matrix[x+2][y+2][i];
		
		//load corner and edges
		if( tx == 0 ){
			for(int i=0;i<D;i++){
				M[tx][ty+2][i] =  matrix[x][y+2][i];
				M[tx+1][ty+2][i] = matrix[x+1][y+2][i];
			}
		}

		if(ty==0){
			for(int i=0;i<D;i++){
				M[tx+2][ty][i] =  matrix[x+2][y][i];
				M[tx+2][ty+1][i] = matrix[x+2][y+1][i];
			}
		}

		if(tx==TILE_WIDTH-1){
			for(int i=0;i<D;i++){
				M[tx+3][ty+2][i] =  matrix[x+3][y+2][i];
				M[tx+4][ty+2][i] = matrix[x+4][y+2][i];
			}
		}

		if(ty== TILE_WIDTH - 1){
			for(int i=0;i<D;i++){
				M[tx+2][ty+3][i] =  matrix[x+2][y+3][i];
				M[tx+2][ty+4][i] = matrix[x+2][y+4][i];
			}
		}

		if(tx==0 && ty==0){
			for(int i=0;i<D;i++){
				M[tx][ty][i] =  matrix[x][y][i];
				M[tx+1][ty][i] = matrix[x+1][y][i];
				M[tx][ty+1][i] = matrix[x][y+1][i];
				M[tx+1][ty+1][i]= matrix[x+1][y+1][i];
			}
		}

		if(tx==0 && ty==TILE_WIDTH-1){
			for(int i=0;i<D;i++){
				M[tx][ty+3][i] =  matrix[x][y+3][i];
				M[tx+1][ty+3][i] = matrix[x+1][y+3][i];
				M[tx][ty+4][i] = matrix[x][y+4][i];
				M[tx+1][ty+4][i]= matrix[x+1][y+4][i];
			}
		}

		if(tx==TILE_WIDTH-1 && ty==0){
			for(int i=0;i<D;i++){
				M[tx+3][ty][i] =  matrix[x+3][y][i];
				M[tx+4][ty][i] = matrix[x+4][y][i];
				M[tx+3][ty+1][i] = matrix[x+3][y+1][i];
				M[tx+4][ty+1][i]= matrix[x+4][y+1][i];
			}
		}

		if(tx==TILE_WIDTH-1 && ty==TILE_WIDTH-1){
			for(int i=0;i<D;i++){
				M[tx+3][ty+3][i] =  matrix[x+3][y+3][i];
				M[tx+4][ty+3][i] = matrix[x+4][y+3][i];
				M[tx+3][ty+4][i] = matrix[x+3][y+4][i];
				M[tx+4][ty+4][i]= matrix[x+4][y+4][i];
			}
		}

	}else{
		for(int i=0;i<D;i++)
			M[tx][ty][i]= matrix[x][y][i];
	}


	__syncthreads();

	//compute each output val

	if(x<OW && y<OW){
		float val=0;
		for(int r=0;r< T;r++){
			for(int c=0;c< T;c++){
				for(int i=0;i<D;i++){
					val+= kernel[k_idx][r][c][i] * M[tx+r][ty+c][i];
				}
			}
		}

		out[ (k_idx*OW*OW)+(x*OW)+y]=val;
	}

	
	//get kernel and apply for your number dot product

}


int main()
{

	unsigned char *matrix=(unsigned char*)malloc(sizeof(unsigned char)*W*W*D);
	float *kernel=(float*)malloc(sizeof(float)*T*T*D*N);
	float *output=(float *)malloc(sizeof(float)*N*OW*OW);


	fillMatrix(matrix);
	fillKernel(kernel);


	unsigned char *Dmatrix;hipMalloc(&Dmatrix,sizeof(unsigned char)*W*W*D);
	float *Dkernel;hipMalloc(&Dkernel,sizeof(float)*N*T*T*D);
	float *Doutput;hipMalloc(&Doutput,sizeof(float)*N*OW*OW);

	hipMemcpy(Dmatrix, matrix, sizeof(unsigned char)*W*W*D,hipMemcpyHostToDevice);
	hipMemcpy(Dkernel, kernel, sizeof(float)*T*T*D*N,hipMemcpyHostToDevice);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	hipEventRecord(start,0);

	//Make your cuda kernel call
	dim3 threads_per_block(TILE_WIDTH,TILE_WIDTH);


	int blocks = ( OW + TILE_WIDTH -1)/TILE_WIDTH;
	int block_dim = blocks*blocks; 
	dim3 blocks_per_grid(blocks,blocks,N);

	Convolve<<<blocks_per_grid,threads_per_block>>>(Dmatrix,Doutput,Dkernel);


	hipDeviceSynchronize();


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(float)*N*OW*OW,hipMemcpyDeviceToHost);

	//Use print_matrix_to_file function only 
	print_matrix_to_file(output);

}
