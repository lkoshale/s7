#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define O_TIlE_SIZE 32

__global__ void Transpose(int *T, int *index, int *O, int *new_stride, int size, int rank, int *stride,int start,int chunk)
{
	int j = blockIdx.x * blockDim.x +threadIdx.x;
	// printf("%d--%d--%d:\n",start,j,size);
	if(start + j < size && j < chunk){
	    int rm = index[j];
	    int f_ind =0;
	    for(int i=0;i<rank;i++){
	        f_ind += rm/stride[i]*new_stride[i];
	        rm = rm%stride[i];
	    }   
		O[f_ind]=T[j];
		
		// printf("%d--%d--%d\n",f_ind,T[j],j);
		
	}
}

int main(int argc, char* argv[])
{
	
	int rank;
	scanf("%d",&rank);	
	int extent[rank];
	int permute[rank];
	int stride[rank];
	int i,size = 1;
	for(i = 0; i < rank; i++) {
		scanf("%d",&extent[i]);
		size *= extent[i];
	}
	stride[rank-1] = 1;
	for(i = rank-2; i >= 0; i--){
		stride[i] = stride[i+1]*extent[i+1];
	}

	for(i = 0; i < rank; i++){
		 scanf("%d",&permute[i]);
	}

	int *tensor=(int*)malloc(sizeof(int)*size);
	int *tensor_index = (int*)malloc(sizeof(int)*size);
	int *output=(int*)malloc(sizeof(int)*size);
	int sparse_index = 0;
	
	//take input
	for(i = 0;i < size; i++){
		int val;
		scanf("%d",&val);
		if(val!=0){
			tensor[sparse_index] = val;
			tensor_index[sparse_index] = i;
			sparse_index++;
		}
	}

	int new_stride[rank];
	int inverse_permute_stride[rank];
	int inverse_permute[rank];
    int new_extent[rank];
    for(int i=0;i<rank;i++){
        new_extent[i] = extent[permute[i]];
    }
    
    for(int i=rank-1;i>=0;i--){
        if(i==rank-1)
            new_stride[i]=1;
        else    
            new_stride[i]= new_extent[i+1]*new_stride[i+1];
        
    }

    for(int i=0;i<rank;i++){
        inverse_permute[permute[i]] = i;
    }
    for(int i=0;i<rank;i++){
        inverse_permute_stride[i] = new_stride[inverse_permute[i]];
	}



	//divide in 1mb of data for each iteration streams
    int chunkSize1 = sparse_index/2;
    int chunkSize2= (sparse_index+2-1)/2;
	
	hipStream_t stream1, stream2;
	hipStreamCreate (&stream1); hipStreamCreate (&stream2);


	int *Dtensor_index1;hipMalloc(&Dtensor_index1,sizeof(int)*chunkSize1);
	int *Dtensor1;hipMalloc(&Dtensor1,sizeof(int)*chunkSize1);

	int *Dtensor_index2;hipMalloc(&Dtensor_index2,sizeof(int)*chunkSize2);
	int *Dtensor2;hipMalloc(&Dtensor2,sizeof(int)*chunkSize2);

	int *Doutput;hipMalloc(&Doutput,sizeof(int)*size);
	int *Dnew_stride;hipMalloc(&Dnew_stride,sizeof(int)*rank);
	int *Dpermute;hipMalloc(&Dpermute,sizeof(int)*rank);
	int *Dstride;hipMalloc(&Dstride,sizeof(int)*rank);

	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	
	int threadsPerBlock = O_TIlE_SIZE;
    int blocksPerGrid1 = (chunkSize1+O_TIlE_SIZE-1)/O_TIlE_SIZE;
    int blocksPerGrid2 = (chunkSize2+O_TIlE_SIZE-1)/O_TIlE_SIZE;

	hipMemcpy(Dpermute, permute, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dstride, stride, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dnew_stride, inverse_permute_stride, sizeof(int)*rank,hipMemcpyHostToDevice);



	
//	hipMemcpy(Dtensor, tensor, sizeof(int)*sparse_index,hipMemcpyHostToDevice);
//	hipMemcpy(Dtensor_index, tensor_index, sizeof(int)*sparse_index,hipMemcpyHostToDevice);
	hipMemset(Doutput,0,sizeof(int)*size);

	hipEventRecord(start,stream1);
	
    hipMemcpyAsync(Dtensor1, tensor, sizeof(int)*chunkSize1,hipMemcpyHostToDevice,stream1);
    hipMemcpyAsync(Dtensor_index1, tensor_index, sizeof(int)*chunkSize1,hipMemcpyHostToDevice,stream1);

    Transpose<<<blocksPerGrid, threadsPerBlock,0,stream1>>>(Dtensor1, Dtensor_index1, Doutput, Dnew_stride,sparse_index, rank, Dstride,0,chunkSize1);

    hipMemcpyAsync(Dtensor2, tensor+chunkSize1, sizeof(int)*chunkSize2,hipMemcpyHostToDevice,stream2);
    hipMemcpyAsync(Dtensor_index2, tensor_index+chunkSize1, sizeof(int)*chunkSize2,hipMemcpyHostToDevice,stream2);

    Transpose<<<blocksPerGrid, threadsPerBlock,0,stream2>>>(Dtensor2, Dtensor_index2, Doutput, Dnew_stride,sparse_index, rank, Dstride,chunkSize1,chunkSize2);

	hipDeviceSynchronize();

	hipEventRecord(stop,stream1);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(int)*size,hipMemcpyDeviceToHost);

	for(i=0;i<size;i++) printf("%d ",output[i]);
}
