#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define O_TIlE_SIZE 32

__global__ void Transpose(int *T, int *index, int *O, int *new_stride, int size, int rank, int *stride,int start,int chunk)
{
	unsigned int j = blockIdx.x * blockDim.x +threadIdx.x;
	//printf("%d--%d--%d:\n",start,j,size);
	if(start + j < size && j < chunk){
	    unsigned int rm = index[j];
	    unsigned int f_ind =0;
	    for(int i=0;i<rank;i++){
	        f_ind += rm/stride[i]*new_stride[i];
	        rm = rm%stride[i];
	    }   
		O[f_ind]=T[j];
		
		//printf("%d--%d--%d\n",f_ind,T[j],j);
		
	}
}

int main(int argc, char* argv[])
{
	
	int rank;
	scanf("%d",&rank);	
	int extent[rank];
	int permute[rank];
	int stride[rank];
	int i;
	unsigned int size = 1;
	for(i = 0; i < rank; i++) {
		scanf("%d",&extent[i]);
		size *= extent[i];
	}
	stride[rank-1] = 1;
	for(i = rank-2; i >= 0; i--){
		stride[i] = stride[i+1]*extent[i+1];
	}

	for(i = 0; i < rank; i++){
		 scanf("%d",&permute[i]);
	}

	int *tensor=(int*)malloc(sizeof(int)*size);
	int *tensor_index = (int*)malloc(sizeof(int)*size);
	int *output=(int*)malloc(sizeof(int)*size);
	unsigned int sparse_index = 0;
	
	//take input
	for(unsigned int i = 0;i < size; i++){
		int val;
		scanf("%d",&val);
		if(val!=0){
			tensor[sparse_index] = val;
			tensor_index[sparse_index] = i;
			sparse_index++;
		}
	}

	int new_stride[rank];
	int inverse_permute_stride[rank];
	int inverse_permute[rank];
    int new_extent[rank];
    for(int i=0;i<rank;i++){
        new_extent[i] = extent[permute[i]];
    }
    
    for(int i=rank-1;i>=0;i--){
        if(i==rank-1)
            new_stride[i]=1;
        else    
            new_stride[i]= new_extent[i+1]*new_stride[i+1];
        
    }

    for(int i=0;i<rank;i++){
        inverse_permute[permute[i]] = i;
    }
    for(int i=0;i<rank;i++){
        inverse_permute_stride[i] = new_stride[inverse_permute[i]];
	}



	//divide in 1mb of data for each iteration streams
	unsigned int chunkSize = 1024*1024;

	// printf("-%d\n",sparse_index);
	if(sparse_index < chunkSize*2){
		chunkSize = (sparse_index+1)/2;
		// printf("if: %d\n",chunkSize);
	}
	
	hipStream_t stream1, stream2;
	hipStreamCreate (&stream1); hipStreamCreate (&stream2);


	int *Dtensor_index1;hipMalloc(&Dtensor_index1,sizeof(int)*chunkSize);
	int *Dtensor1;hipMalloc(&Dtensor1,sizeof(int)*chunkSize);

	int *Dtensor_index2;hipMalloc(&Dtensor_index2,sizeof(int)*chunkSize);
	int *Dtensor2;hipMalloc(&Dtensor2,sizeof(int)*chunkSize);

	int *Doutput;hipMalloc(&Doutput,sizeof(int)*size);
	int *Dnew_stride;hipMalloc(&Dnew_stride,sizeof(int)*rank);
	int *Dpermute;hipMalloc(&Dpermute,sizeof(int)*rank);
	int *Dstride;hipMalloc(&Dstride,sizeof(int)*rank);

	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	
	int threadsPerBlock = O_TIlE_SIZE;
	int blocksPerGrid = (chunkSize+O_TIlE_SIZE-1)/O_TIlE_SIZE;

	hipMemcpy(Dpermute, permute, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dstride, stride, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dnew_stride, inverse_permute_stride, sizeof(int)*rank,hipMemcpyHostToDevice);



	
//	cudaMemcpy(Dtensor, tensor, sizeof(int)*sparse_index,cudaMemcpyHostToDevice);
//	cudaMemcpy(Dtensor_index, tensor_index, sizeof(int)*sparse_index,cudaMemcpyHostToDevice);
	hipMemset(Doutput,0,sizeof(int)*size);


	hipEventRecord(start,stream1);
	
	for(unsigned int j=0;j<sparse_index;j+=2*chunkSize){
		hipMemcpyAsync(Dtensor1, tensor+j, sizeof(int)*chunkSize,hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(Dtensor_index1, tensor_index+j, sizeof(int)*chunkSize,hipMemcpyHostToDevice,stream1);

		Transpose<<<blocksPerGrid, threadsPerBlock,0,stream1>>>(Dtensor1, Dtensor_index1, Doutput, Dnew_stride,sparse_index, rank, Dstride,j,chunkSize);

		if(j+chunkSize>=sparse_index)
			break;

		hipMemcpyAsync(Dtensor2, tensor+j+chunkSize, sizeof(int)*chunkSize,hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(Dtensor_index2, tensor_index+j+chunkSize, sizeof(int)*chunkSize,hipMemcpyHostToDevice,stream2);

		Transpose<<<blocksPerGrid, threadsPerBlock,0,stream2>>>(Dtensor2, Dtensor_index2, Doutput, Dnew_stride,sparse_index, rank, Dstride,j+chunkSize,chunkSize);

	}
	
	hipDeviceSynchronize();

	hipEventRecord(stop,stream1);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(int)*size,hipMemcpyDeviceToHost);

	//for(unsigned int i=0;i<size;i++) printf("%d ",output[i]);

	return 0;
}
