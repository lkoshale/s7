#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>

#define O_TIlE_SIZE 32

__global__ void Transpose(int *T, int *index, int *O, int *new_stride, int size, int rank, int *stride)
{
	int j = blockIdx.x * blockDim.x +threadIdx.x;
	if(j < size){
	    int rm = index[j];
	    int f_ind =0;
	    for(int i=0;i<rank;i++){
	        f_ind += rm/stride[i]*new_stride[i];
	        rm = rm%stride[i];
	    }   
	    O[f_ind]=T[j];
	}
}

int main(int argc, char* argv[])
{
	
	int rank;
	scanf("%d",&rank);	
	int extent[rank];
	int permute[rank];
	int stride[rank];
	int i,size = 1;
	for(i = 0; i < rank; i++) {
		scanf("%d",&extent[i]);
		size *= extent[i];
	}
	stride[rank-1] = 1;
	for(i = rank-2; i >= 0; i--){
		stride[i] = stride[i+1]*extent[i+1];
	}

	for(i = 0; i < rank; i++){
		 scanf("%d",&permute[i]);
	}

	int *tensor=(int*)malloc(sizeof(int)*size);
	int *tensor_index = (int*)malloc(sizeof(int)*size);
	int *output=(int*)malloc(sizeof(int)*size);
	int sparse_index = 0;
	
	//take input
	for(i = 0;i < size; i++){
		int val;
		scanf("%d",&val);
		if(val!=0){
			tensor[sparse_index] = val;
			tensor_index[sparse_index] = i;
			sparse_index++;
		}
	}

	int new_stride[rank];
	int inverse_permute_stride[rank];
	int inverse_permute[rank];
    int new_extent[rank];
    for(int i=0;i<rank;i++){
        new_extent[i] = extent[permute[i]];
    }
    
    for(int i=rank-1;i>=0;i--){
        if(i==rank-1)
            new_stride[i]=1;
        else    
            new_stride[i]= new_extent[i+1]*new_stride[i+1];
        
    }

    for(int i=0;i<rank;i++){
        inverse_permute[permute[i]] = i;
    }
    for(int i=0;i<rank;i++){
        inverse_permute_stride[i] = new_stride[inverse_permute[i]];
	}



	int *Dtensor_index;hipMalloc(&Dtensor_index,sizeof(int)*sparse_index);
	int *Dtensor;hipMalloc(&Dtensor,sizeof(int)*sparse_index);
	int *Doutput;hipMalloc(&Doutput,sizeof(int)*size);
	int *Dnew_stride;hipMalloc(&Dnew_stride,sizeof(int)*rank);
	int *Dpermute;hipMalloc(&Dpermute,sizeof(int)*rank);
	int *Dstride;hipMalloc(&Dstride,sizeof(int)*rank);

	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	
	int threadsPerBlock = O_TIlE_SIZE;
	int blocksPerGrid = (sparse_index+O_TIlE_SIZE-1)/O_TIlE_SIZE;

	

	hipMemcpy(Dpermute, permute, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dstride, stride, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dnew_stride, inverse_permute_stride, sizeof(int)*rank,hipMemcpyHostToDevice);


	hipEventRecord(start,0);
	
	hipMemcpy(Dtensor, tensor, sizeof(int)*sparse_index,hipMemcpyHostToDevice);
	hipMemcpy(Dtensor_index, tensor_index, sizeof(int)*sparse_index,hipMemcpyHostToDevice);
	hipMemset(Doutput,0,sizeof(int)*size);
	
	Transpose<<<blocksPerGrid, threadsPerBlock>>>(Dtensor, Dtensor_index, Doutput, Dnew_stride, sparse_index, rank, Dstride);

	hipDeviceSynchronize();


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(int)*size,hipMemcpyDeviceToHost);


	for(i=0;i<size;i++) printf("%d ",output[i]);
}
