#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define O_TIlE_SIZE 32


//use const memory 
//use shared memory for input burst (same burst)

//try to paramaterize streams

__global__ void Transpose(int *T, int *O, const int* __restrict__ new_stride, int size, int rank,const  int* __restrict__ permute,const int* __restrict__ stride,int start)
{
	int j = blockIdx.x * blockDim.x +threadIdx.x + start;
	if(j < size+start){
	    int rm = j;
	    int f_ind =0;
	    for(int i=0;i<rank;i++){
	        f_ind += rm/stride[i]*new_stride[i];
	        rm = rm%stride[i];
	    }   
	    O[f_ind]=T[j-start];
	}
}

int main(int argc, char* argv[])
{

	int rank = atoi(argv[1]);
	int extent[rank];
	int permute[rank];
	int stride[rank];
	int i,size = 1;
	for(i = 0; i < rank; i++) {
		extent[i] = atoi(argv[i+2]);
		size *= extent[i];
	}
	stride[rank-1] = 1;
	for(i = rank-2; i >= 0; i--){
		stride[i] = stride[i+1]*extent[i+1];
	}
	for(i = 0; i < rank; i++) permute[i] = atoi(argv[i+rank+2]);

    
    //create streams
    hipStream_t stream1, stream2;
    hipStreamCreate (&stream1); hipStreamCreate (&stream2);

    //create two input vectors for two streams
    int s1 = size/2;                    //floor
    int s2 = (size+2-1)/2;              //ciel
    int *tensor1=(int*)malloc(sizeof(int)*s1);
    int *tensor2=(int*)malloc(sizeof(int)*s2);
    
    int *output=(int*)malloc(sizeof(int)*size);

    for(i = 0;i < size; i++) {
		if(i<s1) tensor1[i] = i;
		else tensor2[i-s1]=i;
	}

    //alocate memory on device
    int *Dtensor1;hipMalloc(&Dtensor1,sizeof(int)*s1);
    int *Dtensor2;hipMalloc(&Dtensor2,sizeof(int)*s2);

	int *Doutput;hipMalloc(&Doutput,sizeof(int)*size);
	int *Dnew_stride;hipMalloc(&Dnew_stride,sizeof(int)*rank);
	int *Dpermute;hipMalloc(&Dpermute,sizeof(int)*rank);
	int *Dstride;hipMalloc(&Dstride,sizeof(int)*rank);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	

	int new_stride[rank];
	int inverse_permute_stride[rank];
	int inverse_permute[rank];
    int new_extent[rank];
    for(int i=0;i<rank;i++){
        new_extent[i] = extent[permute[i]];
    }
    
    for(int i=rank-1;i>=0;i--){
        if(i==rank-1)
            new_stride[i]=1;
        else    
            new_stride[i]= new_extent[i+1]*new_stride[i+1];
        
    }

    for(int i=0;i<rank;i++){
        inverse_permute[permute[i]] = i;
    }
    for(int i=0;i<rank;i++){
        inverse_permute_stride[i] = new_stride[inverse_permute[i]];
    }

    int threadsPerBlock = O_TIlE_SIZE;
    int blocksPerGrid1 = (s1+O_TIlE_SIZE-1)/O_TIlE_SIZE;
    int blocksPerGrid2 = (s2+O_TIlE_SIZE-1)/O_TIlE_SIZE;

    hipMemcpy(Dpermute, permute, sizeof(int)*rank,hipMemcpyHostToDevice);
    hipMemcpy(Dstride, stride, sizeof(int)*rank,hipMemcpyHostToDevice);
    hipMemcpy(Dnew_stride, inverse_permute_stride, sizeof(int)*rank,hipMemcpyHostToDevice);

	hipEventRecord(start,stream1);
    //load memory async
    hipMemcpyAsync(Dtensor1, tensor1, sizeof(int)*s1,hipMemcpyHostToDevice,stream1);

	//Dense Kernel
    Transpose<<<blocksPerGrid1, threadsPerBlock,0,stream1>>>(Dtensor1, Doutput, Dnew_stride, s1, rank, Dpermute, Dstride,0);
	
	//second kernel
	hipMemcpyAsync(Dtensor2, tensor2, sizeof(int)*s2,hipMemcpyHostToDevice,stream2);
	Transpose<<<blocksPerGrid2, threadsPerBlock,0,stream2>>>(Dtensor2, Doutput, Dnew_stride, s2, rank, Dpermute, Dstride,s1);

	hipDeviceSynchronize();

	hipEventRecord(stop,stream1);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(int)*size,hipMemcpyDeviceToHost);

	//for(i=0;i<size;i++) printf("%d ",output[i]);
}
