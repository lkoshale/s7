#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define O_TIlE_SIZE 32
#define Rank 3

__global__ void Transpose(int *T, int *O, int *new_stride, int size, int rank, int *permute, int *stride)
{
	int j = blockIdx.x * blockDim.x +threadIdx.x;
	if(j < size){
		int index[Rank];
		int N = rank;
	    int rm = j;
	    for(int i=0;i<N;i++){
	        int id = rm/stride[i];
	        index[i]= id;
	        rm = rm%stride[i];
	    }   
	    int new_index[Rank]; 
	    for(int i=0;i<N;i++){
	        new_index[i]=index[permute[i]];

	    }
	    int f_ind =0;
	    for(int i=0;i<N;i++){
	        f_ind += new_index[i]*new_stride[i];
	    }


	    O[f_ind]=T[j];
	}
}

int main(int argc, char* argv[])
{

	int rank = atoi(argv[1]);
	int extent[rank];
	int permute[rank];
	int stride[rank];
	int i,size = 1;
	for(i = 0; i < rank; i++) {
		extent[i] = atoi(argv[i+2]);
		size *= extent[i];
	}
	stride[rank-1] = 1;
	for(i = rank-2; i >= 0; i--){
		stride[i] = stride[i+1]*extent[i+1];
	}
	for(i = 0; i < rank; i++) permute[i] = atoi(argv[i+rank+2]);

	int *tensor=(int*)malloc(sizeof(int)*size);
	int *output=(int*)malloc(sizeof(int)*size);

	for(i = 0;i < size; i++) tensor[i] = i;

	int *Dtensor;hipMalloc(&Dtensor,sizeof(int)*size);
	int *Doutput;hipMalloc(&Doutput,sizeof(int)*size);
	int *Dnew_stride;hipMalloc(&Dnew_stride,sizeof(int)*rank);
	int *Dpermute;hipMalloc(&Dpermute,sizeof(int)*rank);
	int *Dstride;hipMalloc(&Dstride,sizeof(int)*rank);

	hipMemcpy(Dtensor, tensor, sizeof(int)*size,hipMemcpyHostToDevice);
	hipMemcpy(Dpermute, permute, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dstride, stride, sizeof(int)*rank,hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	hipEventRecord(start,0);
	int threadsPerBlock = O_TIlE_SIZE;
	int blocksPerGrid = (size+O_TIlE_SIZE-1)/O_TIlE_SIZE;

	int new_stride[rank];
    int new_extent[rank];
    for(int i=0;i<rank;i++){
        new_extent[i] = extent[permute[i]];
    }
    
    for(int i=rank-1;i>=0;i--){
        if(i==rank-1)
            new_stride[i]=1;
        else    
            new_stride[i]= new_extent[i+1]*new_stride[i+1];
        
    }
    hipMemcpy(Dnew_stride, new_stride, sizeof(int)*rank,hipMemcpyHostToDevice);
	//Dense Kernel
	Transpose<<<blocksPerGrid, threadsPerBlock>>>(Dtensor, Doutput, Dnew_stride, size, rank, Dpermute, Dstride);

	hipDeviceSynchronize();


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(int)*size,hipMemcpyDeviceToHost);

	for(i=0;i<size;i++) printf("%d ",output[i]);
}
