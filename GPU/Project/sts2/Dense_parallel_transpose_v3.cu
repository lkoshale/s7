#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define O_TIlE_SIZE 32

__global__ void Transpose(int *T, int *O, int *new_stride, unsigned int size, int rank, int *permute, int *stride)
{
	unsigned int j = blockIdx.x * blockDim.x +threadIdx.x;
	if(j < size){
	    unsigned int rm = j;
	    unsigned int f_ind =0;
	    for(int i=0;i<rank;i++){
	        f_ind += rm/stride[i]*new_stride[i];
	        rm = rm%stride[i];
	    }   
	    O[f_ind]=T[j];
	}
}

int main(int argc, char* argv[])
{

	int rank;
	scanf("%d",&rank);
	int extent[rank];
	int permute[rank];
	int stride[rank];
	int i;
	unsigned int size = 1;
	for(i = 0; i < rank; i++) {
		scanf("%d",&extent[i]);
		size *= extent[i];
	}
	stride[rank-1] = 1;
	for(i = rank-2; i >= 0; i--){
		stride[i] = stride[i+1]*extent[i+1];
	}

	for(i = 0; i < rank; i++){
		 scanf("%d",&permute[i]);
	}

	int *tensor=(int*)malloc(sizeof(int)*size);
	int *output=(int*)malloc(sizeof(int)*size);

	for(unsigned int i = 0;i < size; i++){
		scanf("%d",&tensor[i]);
	} 

	int *Dtensor;hipMalloc(&Dtensor,sizeof(int)*size);
	int *Doutput;hipMalloc(&Doutput,sizeof(int)*size);
	int *Dnew_stride;hipMalloc(&Dnew_stride,sizeof(int)*rank);
	int *Dpermute;hipMalloc(&Dpermute,sizeof(int)*rank);
	int *Dstride;hipMalloc(&Dstride,sizeof(int)*rank);

	

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	
	int threadsPerBlock = O_TIlE_SIZE;
	int blocksPerGrid = (size+O_TIlE_SIZE-1)/O_TIlE_SIZE;

	int new_stride[rank];
	int inverse_permute_stride[rank];
	int inverse_permute[rank];
    int new_extent[rank];
    for(int i=0;i<rank;i++){
        new_extent[i] = extent[permute[i]];
    }
    
    for(int i=rank-1;i>=0;i--){
        if(i==rank-1)
            new_stride[i]=1;
        else    
            new_stride[i]= new_extent[i+1]*new_stride[i+1];
        
    }

    for(int i=0;i<rank;i++){
        inverse_permute[permute[i]] = i;
    }
    for(int i=0;i<rank;i++){
        inverse_permute_stride[i] = new_stride[inverse_permute[i]];
	}

	hipMemcpy(Dpermute, permute, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dstride, stride, sizeof(int)*rank,hipMemcpyHostToDevice);
	hipMemcpy(Dnew_stride, inverse_permute_stride, sizeof(int)*rank,hipMemcpyHostToDevice);


	hipEventRecord(start,0);
	
	hipMemcpy(Dtensor, tensor, sizeof(int)*size,hipMemcpyHostToDevice);
	//Dense Kernel
	Transpose<<<blocksPerGrid, threadsPerBlock>>>(Dtensor, Doutput, Dnew_stride, size, rank, Dpermute, Dstride);

	hipDeviceSynchronize();


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(int)*size,hipMemcpyDeviceToHost);

	for(unsigned int i=0;i<size;i++) printf("%d ",output[i]);
}
