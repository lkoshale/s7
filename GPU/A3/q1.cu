#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

#define MAXWORDS 20000


__global__ void kernel1(int* A,int* C,int size){
    __shared__ int hist[20];

    int x = threadIdx.x;
    int idx = (blockIdx.x*blockDim.x)+x;

    if(idx < size){

        //initailize share mem
        if(x<20){
            hist[x]=0;
        }
        __syncthreads();

        int k = A[idx];

        atomicAdd(&hist[k-1],1);

        //sync all threads so to update in global memory
        __syncthreads();

        //first 20 threads write back to global
        if(x <20){
            atomicAdd(&C[x],hist[x]);
        }
        __syncthreads();
    }

}

/*
__global__ void kernel2(int* A,int* C,int size){
    __shared__ int hist[400];
    
    int x = threadIdx.x;
    int idx = (blockIdx.x*blockDim.x)+x;

    if(idx < size){

        //initailize share mem
        if(x<400){
            hist[x]=0;
        }
        __syncthreads();
    
        // get a value and update

        int m = A[idx];
        int n = A[idx+1];

        int h_idx=0;
        if(n/10==0 && m/10==0){
            h_idx = (m*10 + n)-11;
        }
        else if( (n/100==0 && m/10 == 0)
            h_idx = (m*100 + n)
        }
        else if(n/10==0 && m/100=0) {
            h_idx = (m*100)
        }
        else if(n/100==0 && m/100==0){

        }


    }

}
*/

__global__ void kernelfinal(int* A,int* C,int* ind_array,int* C_size,int size,int N,int* lock){
    int x = threadIdx.x;
    int idx = (blockIdx.x*blockDim.x)+x;

    int curidx[5];

    if(idx < size){
        //current window
        for(int i=0;i<N;i++){
            curidx[i]=A[idx+i];
        }

        int i = 0;
        //update val in global if exists
        for(i=0;i<(*C_size);i++){
            //check for index present in window
            int m = 1;
            for(int j=0;j<N;j++){
                if(curidx[j]==ind_array[N*i+j])
                    m = m && 1;    
                else {
                    m = m && 0;
                    break;
                }
            }

            if(m==1){
                atomicAdd(&C[i],1);
                break;
            }
        }

        //add if doesnt exist with locking
        if(i==*C_size){
            //insert at end
            //get lock
            while(atomicCAS(lock,0,1)!=0){
                //loop
            }

            //critical part
            for(int j=0;j<N;j++){
                ind_array[*C_size*N+j]=curidx[j];
            }

            C[*C_size]=1;
            *C_size+=1;

            //unlock
            atomicCAS(lock,1,0);
        }

    }
}


void checkWord(char* word,int* array,int* len){

   int count = 0;
   for(int i=0;i<strlen(word);i++){
        char c= word[i];
        if(c=='-' && count>0){
            array[*len]=count;
            *len+=1;
            count=0;
        }
        else if(c=='.'||c=='!'||c==','||c=='?'||c==';'|c==':'||c=='\''||c=='\"'||c=='('||c==')'||c=='['||c==']'){
            continue;
        }
        else{
            count++;
        }
   }

    if(count>0){
        array[*len]=count;
        *len+=1;
    }


}


void print(int* array,int* len){

    for(int i=0;i<*len;i++){
        printf("%d %d\n",i,array[i]);
    }

}

int main(int argc , char* argv[]){

    int size = sizeof(int)*MAXWORDS;
    int* h_A = (int*)malloc(size);
    
    int N = atoi(argv[1]);    // For calculating N-count-grams
    char *filename = argv[2];  // Filename: shaks.txt


    char curWord[40];   // Take input string into this
    int totalWordCount = 0;
    int* len = (int*)malloc(sizeof(int));
    *len = 0;

    // Count of number of words read
    FILE *ipf = fopen(filename,"r");
    while (fscanf(ipf, "%s",curWord)!=EOF && totalWordCount < MAXWORDS) {
        checkWord(curWord,h_A,len);
        totalWordCount++;
    }

    //size of output histogram 
    //no of combinations can be possible
    int len_C;
    switch(N){
        case 1: len_C=20;
                break;
        case 2: len_C=400;
                    break;
        case 3: len_C=8000;
                    break;
        case 4: len_C= totalWordCount -3;
                break;
        case 5: len_C = totalWordCount -4;
                break;
        
        default: len_C = totalWordCount -N;
    }
    
    //two arrays one for index and one for storage

    int size_C = len_C*sizeof(int);
    int* h_C = (int*)malloc(size_C);
    int* h_index = (int*)malloc(size_C*N);
    int* h_C_size = (int*)malloc(sizeof(int));
    

    // print(h_A,len);

    int* d_A;
    hipMalloc(&d_A,size);
    int* d_C;
    hipMalloc(&d_C,size_C);
    int* d_index;
    hipMalloc(&d_index,size_C*N);
    int* d_C_size;
    hipMalloc(&d_C_size,sizeof(int));
    int* d_lock;
    hipMalloc(&d_lock,sizeof(int));

    //set initail value to 0
    hipMemset(d_C,0,size_C);
    hipMemset(d_C_size,0,sizeof(int));
    hipMemset(d_lock,0,sizeof(int));
    
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    

    int threads_per_block,blocks;

    //dimensions of kernel
    if(N==1){

        //call kernel 1
        threads_per_block = 1024;
        //each block does single computation
        blocks = ( totalWordCount + threads_per_block -1)/threads_per_block;

        kernel1<<<blocks,threads_per_block>>>(d_A,d_C,totalWordCount);
        
        hipMemcpy(h_C,d_C,size_C,hipMemcpyDeviceToHost);
        hipMemcpy(h_index,d_index,size_C*N,hipMemcpyDeviceToHost);
        print(h_C,&len_C);

    }
    else{

        //no use of shared mem
        threads_per_block = 1024;
        int work = totalWordCount - (N-1);

        //1 per op
        blocks = ( work + threads_per_block -1)/threads_per_block;
        
        kernelfinal<<<blocks,threads_per_block>>>(d_A,d_C,d_index,d_C_size,work,N,d_lock);

        hipMemcpy(h_C,d_C,size_C,hipMemcpyDeviceToHost);
        hipMemcpy(h_index,d_index,size_C*N,hipMemcpyDeviceToHost);
        hipMemcpy(h_C_size,d_C_size,sizeof(int),hipMemcpyDeviceToHost);

        for(int i=0;i<*h_C_size;i++){
            for(int j=0;j<N;j++){
                printf("%d ",h_index[i*N+j]);
            }
            printf("%d\n",h_C[i]);
        }
    
    }

    
    printf("%d %d",totalWordCount,blocks);


    //kernel invocation




    //end



}